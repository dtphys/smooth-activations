#include "hip/hip_runtime.h"
#include <vector>
#include <cmath>
#include <iostream>
#include <hip/hip_runtime.h>
#include "smelu.h"

// construct CUDA kernel template for the forward pass
template <typename T>
__global__ void smelu_forward_kernel(const T* input, T* output, const T* alpha, const size_t size) {
    const size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        const T x = input[i];
        output[i] = x >= 0 ? x : alpha[i] * (exp(x) - 1);
    }
}

// construct CUDA kernel template for the backward pass
template <typename T>
__global__ void smelu_backward_kernel(const T* input, const T* grad_output, T* grad_input, const T* alpha, const size_t size) {
    const size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        const T x = input[i];
        grad_input[i] = x >= 0 ? grad_output[i] : grad_output[i] * alpha[i] * exp(x);
    }
}

// construct CUDA kernel template for SmeLU forward pass function
template <typename T>
std::vector<T> SmeLU<T>::forward(const std::vector<T>& input) {
    const size_t size = input.size();
    this->alpha.resize(size);

    T *d_input, *d_output, *d_alpha;
    hipMalloc(&d_input, size * sizeof(T));
    hipMalloc(&d_output, size * sizeof(T));
    hipMalloc(&d_alpha, size * sizeof(T));

    hipMemcpy(d_input, input.data(), size * sizeof(T), hipMemcpyHostToDevice);
    hipMemcpy(d_alpha, this->alpha.data(), size * sizeof(T), hipMemcpyHostToDevice);

    const int block_size = 256;
    const int num_blocks = static_cast<int>((size + block_size - 1) / block_size);
    smelu_forward_kernel<T><<<num_blocks, block_size>>>(d_input, d_output, d_alpha, size);

    std::vector<T> output(size);
    hipMemcpy(output.data(), d_output, size * sizeof(T), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_alpha);

    return output;
}

// construct SmeLU backward pass function
template <typename T>
std::vector<T> SmeLU<T>::backward(const std::vector<T>& input, const std::vector<T>& grad_output) {
    const size_t size = input.size();
    this->alpha.resize(size);

    T *d_input, *d_grad_output, *d_grad_input, *d_alpha;
    hipMalloc(&d_input, size * sizeof(T));
    hipMalloc(&d_grad_output, size * sizeof(T));
    hipMalloc(&d_grad_input, size * sizeof(T));
    hipMalloc(&d_alpha, size * sizeof(T));

    hipMemcpy(d_input, input.data(), size * sizeof(T), hipMemcpyHostToDevice);
    hipMemcpy(d_grad_output, grad_output.data(), size * sizeof(T), hipMemcpyHostToDevice);
    hipMemcpy(d_alpha, this->alpha.data(), size * sizeof(T), hipMemcpyHostToDevice);

    const int block_size = 256;
    const int num_blocks = static_cast<int>((size + block_size - 1) / block_size);
    smelu_backward_kernel<<<num_blocks, block_size>>>(d_input, d_grad_output, d_grad_input, d_alpha, size);

    std::vector<T> grad_input(size);
    hipMemcpy(grad_input.data(), d_grad_input, size * sizeof(T), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_grad_output);
    hipFree(d_grad_input);
    hipFree(d_alpha);

    return grad_input;
}

// init SmeLU templates
template class SmeLU<float>;
template class SmeLU<double>;
